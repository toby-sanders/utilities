
#include <hip/hip_runtime.h>
/*
* SLgetShearlet3DCUDA
*/
__global__ void SLgetShearlet3DCUDA(double2 * pShearlet3D, double * pShearletAbsSqrd,const int pyramid, const double2 * pShearlet2D1, const double2 * pShearlet2D2)  {
	int d1 = blockIdx.x;
	int d2 = blockIdx.y;
	int d3 = threadIdx.x;
	int idx1,idx2,idx3;
	
	idx1 = d1 + (d2 + d3*gridDim.y)*gridDim.x;
	
	if(pyramid == 0  || pyramid == 1){
		idx2 = d1 + d2*gridDim.x;
		idx3 = d3 + d2*blockDim.x;
	}
	else if(pyramid == 2){
		idx2 = d1 + d3*gridDim.x;
		idx3 = d3 + d2*blockDim.x;	
	}
	else{
		idx2 = d1 + d2*gridDim.x;
		idx3 = d1 + d3*gridDim.x;
	}
	
	double re1 = pShearlet2D1[idx2].x;
	double im1 = pShearlet2D1[idx2].y;
	
	double re2 = pShearlet2D2[idx3].x;
	double im2 = pShearlet2D2[idx3].y;
	
	double reSh = re1*re2 - im1*im2;
	double imSh = re1*im2 + re2*im1;
	
	pShearlet3D[idx1].x = reSh;
	pShearlet3D[idx1].y = imSh;
	
	pShearletAbsSqrd[idx1] = reSh*reSh + imSh*imSh;
   
}